#include <iostream>
#include <fstream>
#include <random>
#include <cmath>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>   // Required for CUDA memory management and kernel execution

double G = 6.674 * std::pow(10, -11);
// double G = 1;

struct simulation {
  size_t nbpart;
  
  std::vector<double> mass;

  //position
  std::vector<double> x;
  std::vector<double> y;
  std::vector<double> z;

  //velocity
  std::vector<double> vx;
  std::vector<double> vy;
  std::vector<double> vz;

  //force
  std::vector<double> fx;
  std::vector<double> fy;
  std::vector<double> fz;

  simulation(size_t nb)
    :nbpart(nb), mass(nb),
     x(nb), y(nb), z(nb),
     vx(nb), vy(nb), vz(nb),
     fx(nb), fy(nb), fz(nb) 
  {}
};

void random_init(simulation& s) {
  std::random_device rd;  
  std::mt19937 gen(rd());
  std::uniform_real_distribution<double> dismass(0.9, 1.);
  std::normal_distribution<double> dispos(0., 1.);
  std::normal_distribution<double> disvel(0., 1.);

  for (size_t i = 0; i<s.nbpart; ++i) {
    s.mass[i] = dismass(gen);

    s.x[i] = dispos(gen);
    s.y[i] = dispos(gen);
    s.z[i] = dispos(gen);
    s.z[i] = 0.;
    
    s.vx[i] = disvel(gen);
    s.vy[i] = disvel(gen);
    s.vz[i] = disvel(gen);
    s.vz[i] = 0.;
    s.vx[i] = s.y[i]*1.5;
    s.vy[i] = -s.x[i]*1.5;
  }

  return;
  //normalize velocity (using normalization found on some physics blog)
  double meanmass = 0;
  double meanmassvx = 0;
  double meanmassvy = 0;
  double meanmassvz = 0;
  for (size_t i = 0; i<s.nbpart; ++i) {
    meanmass += s.mass[i];
    meanmassvx += s.mass[i] * s.vx[i];
    meanmassvy += s.mass[i] * s.vy[i];
    meanmassvz += s.mass[i] * s.vz[i];
  }
  for (size_t i = 0; i<s.nbpart; ++i) {
    s.vx[i] -= meanmassvx/meanmass;
    s.vy[i] -= meanmassvy/meanmass;
    s.vz[i] -= meanmassvz/meanmass;
  }
}

void init_solar(simulation& s) {
  enum Planets {SUN, MERCURY, VENUS, EARTH, MARS, JUPITER, SATURN, URANUS, NEPTUNE, MOON};
  s = simulation(10);

  s.mass[SUN] = 1.9891 * std::pow(10, 30);
  s.mass[MERCURY] = 3.285 * std::pow(10, 23);
  s.mass[VENUS] = 4.867 * std::pow(10, 24);
  s.mass[EARTH] = 5.972 * std::pow(10, 24);
  s.mass[MARS] = 6.39 * std::pow(10, 23);
  s.mass[JUPITER] = 1.898 * std::pow(10, 27);
  s.mass[SATURN] = 5.683 * std::pow(10, 26);
  s.mass[URANUS] = 8.681 * std::pow(10, 25);
  s.mass[NEPTUNE] = 1.024 * std::pow(10, 26);
  s.mass[MOON] = 7.342 * std::pow(10, 22);

  // Positions (in meters) and velocities (in m/s)
  double AU = 1.496 * std::pow(10, 11); // Astronomical Unit

  s.x = {0, 0.39*AU, 0.72*AU, 1.0*AU, 1.52*AU, 5.20*AU, 9.58*AU, 19.22*AU, 30.05*AU, 1.0*AU + 3.844*std::pow(10, 8)};
  s.y = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  s.z = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  s.vx = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  s.vy = {0, 47870, 35020, 29780, 24130, 13070, 9680, 6800, 5430, 29780 + 1022};
  s.vz = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
}

/*  //old function

/ Astronomical Unit
void update_force(simulation& s, size_t from, size_t to) {
  double softening = .1;
  double dist_sq = std::pow(s.x[from]-s.x[to],2)
    + std::pow(s.y[from]-s.y[to],2)
    + std::pow(s.z[from]-s.z[to],2);
  double F = G * s.mass[from]*s.mass[to]/(dist_sq+softening); //that the strength of the force

  //direction
  double dx = s.x[from]-s.x[to];
  double dy = s.y[from]-s.y[to];
  double dz = s.z[from]-s.z[to];
  double norm = std::sqrt(dx*dx+dy*dy+dz*dz);
  
  dx = dx/norm;
  dy = dy/norm;
  dz = dz/norm;

  //apply force
  s.fx[to] += dx*F;
  s.fy[to] += dy*F;
  s.fz[to] += dz*F;
}

void reset_force(simulation& s) {
  for (size_t i=0; i<s.nbpart; ++i) {
    s.fx[i] = 0.;
    s.fy[i] = 0.;
    s.fz[i] = 0.;
  }
}

void apply_force(simulation& s, size_t i, double dt) {
  s.vx[i] += s.fx[i]/s.mass[i]*dt;
  s.vy[i] += s.fy[i]/s.mass[i]*dt;
  s.vz[i] += s.fz[i]/s.mass[i]*dt;
}

void update_position(simulation& s, size_t i, double dt) {
  s.x[i] += s.vx[i]*dt;
  s.y[i] += s.vy[i]*dt;
  s.z[i] += s.vz[i]*dt;
}

*/


void dump_state(simulation& s) {
  std::cout<<s.nbpart<<'\t';
  for (size_t i=0; i<s.nbpart; ++i) {
    std::cout<<s.mass[i]<<'\t';
    std::cout<<s.x[i]<<'\t'<<s.y[i]<<'\t'<<s.z[i]<<'\t';
    std::cout<<s.vx[i]<<'\t'<<s.vy[i]<<'\t'<<s.vz[i]<<'\t';
    std::cout<<s.fx[i]<<'\t'<<s.fy[i]<<'\t'<<s.fz[i]<<'\t';
  }
  std::cout<<'\n';
}

void load_from_file(simulation& s, std::string filename) {
  std::ifstream in (filename);
  size_t nbpart;
  in>>nbpart;
  s = simulation(nbpart);
  for (size_t i=0; i<s.nbpart; ++i) {
    in>>s.mass[i];
    in >>  s.x[i] >>  s.y[i] >>  s.z[i];
    in >> s.vx[i] >> s.vy[i] >> s.vz[i];
    in >> s.fx[i] >> s.fy[i] >> s.fz[i];
  }
  if (!in.good())
    throw "kaboom";
}

// CUDA kernel, compute gravitational forces
__global__ void compute_forces_kernel(
  size_t nbpart,
  double G,
  const double* mass,
  const double* x, const double* y, const double* z,
  double* fx, double* fy, double* fz
) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  double xi = x[i], yi = y[i], zi = z[i];
  double m_i = mass[i];
  double fxi = 0., fyi = 0., fzi = 0.;

  for (int j = 0; j < nbpart; ++j) {
    if (i == j) continue;
    double dx = x[j] - xi;
    double dy = y[j] - yi;
    double dz = z[j] - zi;
    double dist_sq = dx*dx + dy*dy + dz*dz + 0.1;
    double dist = sqrt(dist_sq);
    double F = G * m_i * mass[j] / dist_sq;

    fxi += F * dx / dist;
    fyi += F * dy / dist;
    fzi += F * dz / dist;
  }

  fx[i] = fxi;
  fy[i] = fyi;
  fz[i] = fzi;
}

// CUDA kernel,update velocity and position
__global__ void update_positions_kernel(
  size_t nbpart,
  double dt,
  const double* mass,
  double* x, double* y, double* z,
  double* vx, double* vy, double* vz,
  const double* fx, const double* fy, const double* fz
) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbpart) return;

  vx[i] += fx[i] / mass[i] * dt;
  vy[i] += fy[i] / mass[i] * dt;
  vz[i] += fz[i] / mass[i] * dt;

  x[i] += vx[i] * dt;
  y[i] += vy[i] * dt;
  z[i] += vz[i] * dt;
}


// main
int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr
      <<"usage: "<<argv[0]<<" <input> <dt> <nbstep> <printevery> <blocksize>\n"
      <<"input can be:\n"
      <<"a number (random initialization)\n"
      <<"planet (initialize with solar system)\n"
      <<"a filename (load from file in singleline tsv)\n";
    return -1;
  }

  double dt = std::atof(argv[2]);  //in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);
  int blockSize = std::atoi(argv[5]);

  simulation s(1);

  //parse command line
  {
    size_t nbpart = std::atol(argv[1]); //return 0 if not a number
    if ( nbpart > 0) {
      s = simulation(nbpart);
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
        init_solar(s);
      } else{
        load_from_file(s, inputparam);
      }
    }    
  }

  size_t N = s.nbpart, size = N * sizeof(double);
  double *d_mass, *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_fx, *d_fy, *d_fz;

  // allocate GPU memory (required by assignment, use cudaMalloc)
  hipMalloc(&d_mass, size);
  hipMalloc(&d_x, size); hipMalloc(&d_y, size); hipMalloc(&d_z, size);
  hipMalloc(&d_vx, size); hipMalloc(&d_vy, size); hipMalloc(&d_vz, size);
  hipMalloc(&d_fx, size); hipMalloc(&d_fy, size); hipMalloc(&d_fz, size);

  // copy data from host to device (as instructed in assignment use cudaMemcpy)
  hipMemcpy(d_mass, s.mass.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_x, s.x.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, s.y.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_z, s.z.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_vx, s.vx.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_vy, s.vy.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_vz, s.vz.data(), size, hipMemcpyHostToDevice);

  int gridSize = (N + blockSize - 1) / blockSize;

  for (size_t step = 0; step < nbstep; ++step) {
    compute_forces_kernel<<<gridSize, blockSize>>>(
      N, G, d_mass, d_x, d_y, d_z, d_fx, d_fy, d_fz
    );
    update_positions_kernel<<<gridSize, blockSize>>>(
      N, dt, d_mass, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_fx, d_fy, d_fz
    );

    if (step % printevery == 0) {
      hipMemcpy(s.x.data(), d_x, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.y.data(), d_y, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.z.data(), d_z, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.vx.data(), d_vx, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.vy.data(), d_vy, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.vz.data(), d_vz, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.fx.data(), d_fx, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.fy.data(), d_fy, size, hipMemcpyDeviceToHost);
      hipMemcpy(s.fz.data(), d_fz, size, hipMemcpyDeviceToHost);
      dump_state(s);
    }
  }

  hipFree(d_mass); hipFree(d_x); hipFree(d_y); hipFree(d_z);
  hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
  hipFree(d_fx); hipFree(d_fy); hipFree(d_fz);

  return 0;
}

/*  //old main(old man? lol)
int main(int argc, char* argv[]) {
  if (argc != 5) {
    std::cerr
      <<"usage: "<<argv[0]<<" <input> <dt> <nbstep> <printevery>"<<"\n"
      <<"input can be:"<<"\n"
      <<"a number (random initialization)"<<"\n"
      <<"planet (initialize with solar system)"<<"\n"
      <<"a filename (load from file in singleline tsv)"<<"\n";
    return -1;
  }
  
  double dt = std::atof(argv[2]); //in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);
  
  
  simulation s(1);

  //parse command line
  {
    size_t nbpart = std::atol(argv[1]); //return 0 if not a number
    if ( nbpart > 0) {
      s = simulation(nbpart);
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
	init_solar(s);
      } else{
	load_from_file(s, inputparam);
      }
    }    
  }

  
  for (size_t step = 0; step< nbstep; step++) {
    if (step %printevery == 0)
      dump_state(s);
  
    reset_force(s);
    for (size_t i=0; i<s.nbpart; ++i)
      for (size_t j=0; j<s.nbpart; ++j)
	if (i != j)
	  update_force(s, i, j);

    for (size_t i=0; i<s.nbpart; ++i) {
      apply_force(s, i, dt);
      update_position(s, i, dt);
    }
  }
  
  //dump_state(s);  


  return 0;
}

*/
